#include "hip/hip_runtime.h"

/*
 *  Copyright (C) 2013 Petr Klapetek
 *  E-mail: klapetek@gwyddion.net.
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111 USA
 */


/*  gpu_kernels.cu : 
 *  all the algorithms that run on GPU and are compied with nvcc
 *  FIXME: split this into more files
 */

#if defined (_WIN32) && defined (_MSC_VER)
#include "../config_win.h"
#else
#include "../config.h"
#endif
#ifdef UCUDA

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "constants.h"
//#include <cutil.h>
#include <hip/hip_runtime_api.h>
//#include <multithreading.h>

extern "C" {
#include "gpu_kernels.h"
}

#define CU_PI 3.14159265358979323846

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))

/*values in the main field arrays*/
#define CAT(i, j) ((j)*xres + (i))

/*values stored for Liao boundary*/
#define CLX(off, i)  (off*xres + (i)) 
#define CLXP(off, i) ((off + 1)*xres + (i))
#define CLY(off, j)  (off*yres + (j)) 
#define CLYP(off, j) ((off + 1)*yres + (j))

/*various constants to help searching in parameter arrays*/
#define EX 0
#define EY 2
#define EZ 4
#define HX 6
#define HY 8
#define HZ 10

#define I0 0
#define I1 1
#define J0 2
#define J1 3
#define K0 4
#define K1 5
#define SKIPI0 6
#define SKIPI0_JMIN 7
#define SKIPI0_KMIN 8
#define SKIPI0_JMAX 9
#define SKIPI0_KMAX 10
#define SKIPIN 11
#define SKIPIN_JMIN 12
#define SKIPIN_KMIN 13
#define SKIPIN_JMAX 14
#define SKIPIN_KMAX 15

#define SKIPJ0 16
#define SKIPJ0_IMIN 17
#define SKIPJ0_KMIN 18
#define SKIPJ0_IMAX 19
#define SKIPJ0_KMAX 20
#define SKIPJN 21
#define SKIPJN_IMIN 22
#define SKIPJN_KMIN 23
#define SKIPJN_IMAX 24
#define SKIPJN_KMAX 25

#define SKIPK0 26
#define SKIPK0_IMIN 27
#define SKIPK0_JMIN 28
#define SKIPK0_IMAX 29
#define SKIPK0_JMAX 30
#define SKIPKN 31
#define SKIPKN_IMIN 32
#define SKIPKN_JMIN 33
#define SKIPKN_IMAX 34
#define SKIPKN_JMAX 35

#define BNDX0 0
#define BNDXN 1
#define BNDY0 2
#define BNDYN 3
#define BNDZ0 4
#define BNDZN 5

#define NPNTS 36
#define PNPNTS 20
#define PNDIV 200

#define GETEPS(i) (18*i + SV_GM_EPSILON)
#define GETMU(i) (18*i + SV_GM_MU)
#define GETSIGMA(i) (18*i + SV_GM_SIGMA)
#define GETSIGAST(i) (18*i + SV_GM_SIGAST)
#define GETDRUDEO(i) (18*i + SV_GM_DRUDE_OMEGA_P)
#define GETDRUDEN(i) (18*i + SV_GM_DRUDE_NU)
#define GETCPA0(i) (18*i + SV_GM_CP3_A0)
#define GETCPA1(i) (18*i + SV_GM_CP3_A1)
#define GETCPA2(i) (18*i + SV_GM_CP3_A2)
#define GETCPPHI0(i) (18*i + SV_GM_CP3_PHI0)
#define GETCPPHI1(i) (18*i + SV_GM_CP3_PHI1)
#define GETCPPHI2(i) (18*i + SV_GM_CP3_PHI2)
#define GETCPOMEGA0(i) (18*i + SV_GM_CP3_OMEGA0)
#define GETCPOMEGA1(i) (18*i + SV_GM_CP3_OMEGA1)
#define GETCPOMEGA2(i) (18*i + SV_GM_CP3_OMEGA2)
#define GETCPGAMMA0(i) (18*i + SV_GM_CP3_GAMMA0)
#define GETCPGAMMA1(i) (18*i + SV_GM_CP3_GAMMA1)
#define GETCPGAMMA2(i) (18*i + SV_GM_CP3_GAMMA2)

#define MU_0 1.256637061435917295e-6
#define EPSILON_0 8.854187817620389850e-12


#define MEM_N 0
#define LS 299792458.0
#define JCPI 0.0795774715 
/*(1/4Pi)*/

#define SV_BOUNDARY_CPML 3

__device__ float k_dcomp(int i, int j, int k, int xres, int yres, int zres, float theta, float phi,
                         int i0, int i1, int j0, int j1, int k0, int k1);

__device__ float k_gex(float field, float theta, float phi, float psi);
__device__ float k_gey(float field, float theta, float phi, float psi);
__device__ float k_gez(float field, float theta, float phi, float psi);
__device__ float k_ghx(float field, float theta, float phi, float psi);
__device__ float k_ghy(float field, float theta, float phi, float psi);
__device__ float k_ghz(float field, float theta, float phi, float psi);
__device__ float get_dval(float *line, int res, float x);


/*vacuum or any material given by voxel-by-voxel set of optical parameters*/

__global__  void
eKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         float *d_epsilon, float *d_mu, float *d_sigma, float *d_sigast, int xres, int yres, int tmmode,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    float ca, cb;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i<=0 || j<=0) return;
    if (i>=(xres) || j>=(yres)) return;

    pos = CAT(i, j);

    ca  = (1 - d_sigma[pos]*dt/2/d_epsilon[pos])/(1 + d_sigma[pos]*dt/2/d_epsilon[pos]);
    cb  = (dt/d_epsilon[pos]/dx)/(1 + d_sigma[pos]*dt/2/d_epsilon[pos]);


    if (tmmode) {
        
        d_ez[pos] = ca*d_ez[pos] + cb*((d_hy[pos] - d_hy[CAT(i-1, j)]) -
                            (d_hx[pos] - d_hx[CAT(i, j-1)]));
    }
    else {
        d_ex[pos] = ca*d_ex[pos] + cb*(d_hz[pos] - d_hz[CAT(i, j-1)]);

        d_ey[pos] = ca*d_ey[pos] - cb*(d_hz[pos] - d_hz[CAT(i-1, j)]);
    }

}

/*any material including tabulated material (voxel-by-voxel set of tabulated material parameters)*/
__global__  void
eKernel_tab(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         float *d_epsilon, float *d_mu, float *d_sigma, float *d_sigast, 
         int *d_mat, int *d_mattype, float *d_mattab, int nmat, int matmode,
         int xres, int yres, int tmmode,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    float ca, cb, sigma, epsilon;
    long int pos;
    int mattype_xm, mattype_ym, mattype;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i<=0 || j<=0) return;
    if (i>=(xres) || j>=(yres)) return;

    pos = CAT(i, j);

    if (i>0 && j>0) {
	    mattype_xm = d_mattype[d_mat[CAT(i-1, j)]];
	    mattype_ym = d_mattype[d_mat[CAT(i, j-1)]];
    }

    mattype = d_mattype[d_mat[pos]];

    if (d_mat[pos]==0 && !(matmode==0 || matmode==2)) { //= mattype 0 = linear material given pixel by pixel
	    sigma = d_sigma[pos];
	    epsilon = d_epsilon[pos];
    } else if (mattype==1) { //tabulated linear material, here should be also the cp3 and drude option
	    sigma = d_mattab[GETSIGMA(d_mat[pos])];
	    epsilon = d_mattab[GETEPS(d_mat[pos])];
    } else {
	    sigma = 0;
	    epsilon = EPSILON_0;
    }
    ca  = (1 - sigma*dt/2/epsilon)/(1 + sigma*dt/2/epsilon);
    cb  = (dt/epsilon/dx)/(1 + sigma*dt/2/epsilon);

    if (tmmode) {
        
        d_ez[pos] = ca*d_ez[pos] + cb*((d_hy[pos] - d_hy[CAT(i-1, j)]) -
                            (d_hx[pos] - d_hx[CAT(i, j-1)]));
    }
    else {
        d_ex[pos] = ca*d_ex[pos] + cb*(d_hz[pos] - d_hz[CAT(i, j-1)]);

        d_ey[pos] = ca*d_ey[pos] - cb*(d_hz[pos] - d_hz[CAT(i-1, j)]);
    }
  
    /*PEC treatment*/
    if (i>0 && j>0) {
	    if ((mattype_xm!=10 && mattype==10) || (mattype_xm==10 && mattype!=10))
	    {
		    d_ey[pos] = 0;
		    d_ez[pos] = 0;
	    }

	    if ((mattype_ym!=10 && mattype==10) || (mattype_ym==10 && mattype!=10))
	    {
		    d_ex[pos] = 0;
		    d_ez[pos] = 0;
	    }
    }


}

__global__  void
eKernel_none(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         int xres, int yres, int tmmode,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    float cb;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i<=0 || j<=0) return;
    if (i>=(xres) || j>=(yres)) return;

    pos = CAT(i, j);
    cb  = dt/EPSILON_0/dx;

    if (tmmode) {
        
        d_ez[pos] = d_ez[pos] + cb*((d_hy[pos] - d_hy[CAT(i-1, j)]) -
                            (d_hx[pos] - d_hx[CAT(i, j-1)]));
    }
    else {
        d_ex[pos] = d_ex[pos] + cb*(d_hz[pos] - d_hz[CAT(i, j-1)]);

        d_ey[pos] = d_ey[pos] - cb*(d_hz[pos] - d_hz[CAT(i-1, j)]);
    }

}

/*any linear material given voxel-by-voxel. Note that hKernel_tab (for tabulated magnetic material) is not implemented now*/
__global__  void
hKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         float *d_epsilon, float *d_mu, float *d_sigma, float *d_sigast, int xres, int yres, int tmmode,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    float da, db;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i<0 || j<0) return;
    if (i>=(xres-1) || j>=(yres-1)) return;

    pos = CAT(i, j);

    da  = (1 - d_sigast[pos]*dt/2/d_mu[pos])/(1 + d_sigast[pos]*dt/2/d_mu[pos]);
    db  = (dt/d_mu[pos]/dx)/(1 + d_sigast[pos]*dt/2/d_mu[pos]);

    if (tmmode) {

	    d_hx[pos] = da*d_hx[pos] - db*(d_ez[CAT(i, j+1)] - d_ez[pos]);

	    d_hy[pos] = da*d_hy[pos] + db*(d_ez[CAT(i+1, j)] - d_ez[pos]);
    } else {

	    d_hz[pos] = da*d_hz[pos] + db*((d_ex[CAT(i, j+1)] - d_ex[pos]) -
			    (d_ey[CAT(i+1, j)] - d_ey[pos]));
    }


}

/*vaccum only*/
__global__  void
hKernel_none(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         int xres, int yres, int tmmode,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    float db;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i<0 || j<0) return;
    if (i>=(xres-1) || j>=(yres-1)) return;

    pos = CAT(i, j);

    db  = dt/MU_0/dx;

    if (tmmode) {

	    d_hx[pos] = d_hx[pos] - db*(d_ez[CAT(i, j+1)] - d_ez[pos]);

	    d_hy[pos] = d_hy[pos] + db*(d_ez[CAT(i+1, j)] - d_ez[pos]);
    } else {

	    d_hz[pos] = d_hz[pos] + db*((d_ex[CAT(i, j+1)] - d_ex[pos]) -
			    (d_ey[CAT(i+1, j)] - d_ey[pos]));
    }
}

/*copy data for absorbing boundary condition*/
__global__  void
liaocpyKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
         float *d_x0, float *d_xn, float *d_y0, float *d_yn, 
         int xres, int yres, int* bnds,
         float dx, float dy, float dt, int dir)
{
    int i, j;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    pos = CAT(i, j);

    if (bnds[BNDX0] == 2) {
        if (i==0) {
            d_x0[CLX(EX, j)] = d_ex[pos];
            d_x0[CLX(EY, j)] = d_ey[pos];
            d_x0[CLX(EZ, j)] = d_ez[pos];
            d_x0[CLX(HX, j)] = d_hx[pos];
            d_x0[CLX(HY, j)] = d_hy[pos];
            d_x0[CLX(HZ, j)] = d_hz[pos];
        } else if (i==1) {
            d_x0[CLXP(EX, j)] = d_ex[pos];
            d_x0[CLXP(EY, j)] = d_ey[pos];
            d_x0[CLXP(EZ, j)] = d_ez[pos];
            d_x0[CLXP(HX, j)] = d_hx[pos];
            d_x0[CLXP(HY, j)] = d_hy[pos];
            d_x0[CLXP(HZ, j)] = d_hz[pos];
        }
    }

    if (bnds[BNDXN] == 2) {
        if (i==(xres-1)) {
            d_xn[CLX(EX, j)] = d_ex[pos];
            d_xn[CLX(EY, j)] = d_ey[pos];
            d_xn[CLX(EZ, j)] = d_ez[pos];
            d_xn[CLX(HX, j)] = d_hx[pos];
            d_xn[CLX(HY, j)] = d_hy[pos];
            d_xn[CLX(HZ, j)] = d_hz[pos];
        } else if (i==(xres-2)) {
            d_xn[CLXP(EX, j)] = d_ex[pos];
            d_xn[CLXP(EY, j)] = d_ey[pos];
            d_xn[CLXP(EZ, j)] = d_ez[pos];
            d_xn[CLXP(HX, j)] = d_hx[pos];
            d_xn[CLXP(HY, j)] = d_hy[pos];
            d_xn[CLXP(HZ, j)] = d_hz[pos];
        }
    }

    if (bnds[BNDY0] == 2) {
        if (j==0) {
            d_y0[CLY(EX, i)] = d_ex[pos];
            d_y0[CLY(EY, i)] = d_ey[pos];
            d_y0[CLY(EZ, i)] = d_ez[pos];
            d_y0[CLY(HX, i)] = d_hx[pos];
            d_y0[CLY(HY, i)] = d_hy[pos];
            d_y0[CLY(HZ, i)] = d_hz[pos];
        } else if (j==1) {
            d_y0[CLYP(EX, i)] = d_ex[pos];
            d_y0[CLYP(EY, i)] = d_ey[pos];
            d_y0[CLYP(EZ, i)] = d_ez[pos];
            d_y0[CLYP(HX, i)] = d_hx[pos];
            d_y0[CLYP(HY, i)] = d_hy[pos];
            d_y0[CLYP(HZ, i)] = d_hz[pos];
        }
    }

    if (bnds[BNDYN] == 2) {
        if (j==(yres-1)) {
            d_yn[CLY(EX, i)] = d_ex[pos];
            d_yn[CLY(EY, i)] = d_ey[pos];
            d_yn[CLY(EZ, i)] = d_ez[pos];
            d_yn[CLY(HX, i)] = d_hx[pos];
            d_yn[CLY(HY, i)] = d_hy[pos];
            d_yn[CLY(HZ, i)] = d_hz[pos];
        } else if (j==(yres-2)) {
            d_yn[CLYP(EX, i)] = d_ex[pos];
            d_yn[CLYP(EY, i)] = d_ey[pos];
            d_yn[CLYP(EZ, i)] = d_ez[pos];
            d_yn[CLYP(HX, i)] = d_hx[pos];
            d_yn[CLYP(HY, i)] = d_hy[pos];
            d_yn[CLYP(HZ, i)] = d_hz[pos];
        }
    }
}


/*run the absorbing boundary conditon*/
__global__  void
liaorunKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz, float *d_epsilon,
         float *d_x0, float *d_xn, float *d_y0, float *d_yn, 
         int xres, int yres, int* bnds, int matmode,
         float dx, float dy, float dt, int dir)
{
    float lssx, lssy, ind;
    float mx, my;
    int i, j;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    pos = CAT(i, j);

    if (matmode==1 || matmode == 3)
        ind = sqrt(d_epsilon[pos]/EPSILON_0);
    else ind = 1;

    lssx = (dt*LS/ind-dx)/(dt*LS/ind+dx);
    lssy = (dt*LS/ind-dy)/(dt*LS/ind+dy);
    mx = (float)lssx;
    my = (float)lssy;

    if (bnds[BNDX0] == 2) {
        if (i==0) {
            d_ex[pos] = d_x0[CLXP(EX, j)] + mx*(d_ex[CAT(i+1, j)] - d_x0[CLX(EX, j)]);
            d_ey[pos] = d_x0[CLXP(EY, j)] + mx*(d_ey[CAT(i+1, j)] - d_x0[CLX(EY, j)]);
            d_ez[pos] = d_x0[CLXP(EZ, j)] + mx*(d_ez[CAT(i+1, j)] - d_x0[CLX(EZ, j)]);
        }
    }
    if (bnds[BNDXN] == 2) {
        if (i==(xres-1)) {
            d_ex[pos] = d_xn[CLXP(EX, j)] + mx*(d_ex[CAT(i-1, j)] - d_xn[CLX(EX, j)]);
            d_ey[pos] = d_xn[CLXP(EY, j)] + mx*(d_ey[CAT(i-1, j)] - d_xn[CLX(EY, j)]);
            d_ez[pos] = d_xn[CLXP(EZ, j)] + mx*(d_ez[CAT(i-1, j)] - d_xn[CLX(EZ, j)]);
            d_hx[pos] = d_xn[CLXP(HX, j)] + mx*(d_hx[CAT(i-1, j)] - d_xn[CLX(HX, j)]);
            d_hy[pos] = d_xn[CLXP(HY, j)] + mx*(d_hy[CAT(i-1, j)] - d_xn[CLX(HY, j)]);
            d_hz[pos] = d_xn[CLXP(HZ, j)] + mx*(d_hz[CAT(i-1, j)] - d_xn[CLX(HZ, j)]);
        }
    }

    if (bnds[BNDY0] == 2) {
        if (j==0) {
            d_ex[pos] = d_y0[CLYP(EX, i)] + my*(d_ex[CAT(i, j+1)] - d_y0[CLY(EX, i)]);
            d_ey[pos] = d_y0[CLYP(EY, i)] + my*(d_ey[CAT(i, j+1)] - d_y0[CLY(EY, i)]);
            d_ez[pos] = d_y0[CLYP(EZ, i)] + my*(d_ez[CAT(i, j+1)] - d_y0[CLY(EZ, i)]);
        }
    }
    if (bnds[BNDYN] == 2) {
        if (j==(yres-1)) {
            d_ex[pos] = d_yn[CLYP(EX, i)] + my*(d_ex[CAT(i, j-1)] - d_yn[CLY(EX, i)]);
            d_ey[pos] = d_yn[CLYP(EY, i)] + my*(d_ey[CAT(i, j-1)] - d_yn[CLY(EY, i)]);
            d_ez[pos] = d_yn[CLYP(EZ, i)] + my*(d_ez[CAT(i, j-1)] - d_yn[CLY(EZ, i)]);
            d_hx[pos] = d_yn[CLYP(HX, i)] + my*(d_hx[CAT(i, j-1)] - d_yn[CLY(HX, i)]);
            d_hy[pos] = d_yn[CLYP(HY, i)] + my*(d_hy[CAT(i, j-1)] - d_yn[CLY(HY, i)]);
            d_hz[pos] = d_yn[CLYP(HZ, i)] + my*(d_hz[CAT(i, j-1)] - d_yn[CLY(HZ, i)]);
        }
    }
}


/*electric field point source*/
__global__  void
srcepointKernel(float *d_ex, float *d_ey, float *d_ez, int xres, int yres, int ipos, int jpos,
         float ex, float ey, float ez, int dir)
{
    int i, j;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i==ipos && j==jpos) {  
       pos = CAT(i, j);
       if (ex != 0) d_ex[pos] = ex;
       if (ey != 0) d_ey[pos] = ey;
       if (ez != 0) d_ez[pos] = ez;
    }

}

/*magnetic field point source*/
__global__  void
srchpointKernel(float *d_hx, float *d_hy, float *d_hz, int xres, int yres, int ipos, int jpos,
         float hx, float hy, float hz, int dir)
{
    int i, j;
    long int pos;

    i = threadIdx.x;
    j = blockIdx.x;

    if (i==ipos && j==jpos) {  
       pos = CAT(i, j);
       if (hx != 0) d_hx[pos] = hx;
       if (hy != 0) d_hy[pos] = hy;
       if (hz != 0) d_hz[pos] = hz;
    }
}

/*set of functions for TSF source*/
/*
__inline__ __device__  float
k_dist(float x1, float y1, float z1, float x2, float y2, float z2)
{
    return sqrtf((x1 - x2)*(x1 - x2)
                + (y1 - y2)*(y1 - y2)
                + (z1 - z2)*(z1 - z2));
}

__inline__ __device__ float 
k_angvec(float x1, float y1, float z1, float x2, float y2, float z2)
{
    return ((x1*x2 + y1*y2 + z1*z2)/(sqrtf((float)(x1*x1 + y1*y1 + z1*z1)*(x2*x2 + y2*y2 + z2*z2))));
}

__device__ float 
k_angle(float x1, float y1, float z1, float x2, float y2, float z2, int side) 
{
    if (side == 0) return k_angvec(x2 - x1, y2 - y1, z2 - z1, 0, -1, 0);
    else if (side == 1) return k_angvec(x2 - x1, y2 - y1, z2 - z1, 0, 1, 0);
    else if (side == 2) return k_angvec(x2 - x1, y2 - y1, z2 - z1, -1, 0, 0);
    else if (side == 3) return k_angvec(x2 - x1, y2 - y1, z2 - z1, 1, 0, 0);
    else if (side == 4) return k_angvec(x2 - x1, y2 - y1, z2 - z1, 0, 0, -1);
    else return k_angvec(x2 - x1, y2 - y1, z2 - z1, 0, 0, 1);
}


*/

/*
__global__  void
tsfjstepKernel(float *d_jpool_e, float *d_jpool_h, float *d_jpvals,
            float *d_jpool_epsilon, float *d_jpool_mu,
            float *d_jpool_sigma, float *d_jpool_sigast, 
            float dx, float dt, float e, int n)
{
        int i;
        //copybound
        float de0, de1, dh0, dh1, den, dem, dhn, dhm;
        float dAngleMult=1;
        float dMulth=dt/(MU_0*dx)/dAngleMult;
        float dMulte=dt/(EPSILON_0*dx)/dAngleMult;         

        de0 = d_jpvals[0];
        de1 = d_jpvals[1];
        dh0 = d_jpvals[2];
        dh1 = d_jpvals[3];
        den = d_jpvals[4];
        dem = d_jpvals[5];
        dhn = d_jpvals[6];
        dhm = d_jpvals[7];

        //ystep_e
        for (i=1; i<(n); i++)
           d_jpool_e[i] += 1.0/(d_jpool_epsilon[i])*dMulte*(d_jpool_h[i-1] - d_jpool_h[i]);

        //bound
        d_jpool_e[0] = de1 + (dt*LS-dx)/(dt*LS+dx)*(d_jpool_e[1] - de0);
        d_jpool_h[0] = dh1 + (dt*LS-dx)/(dt*LS+dx)*(d_jpool_h[1] - dh0);
        d_jpool_e[n-1] = dem + (dt*LS-dx)/(dt*LS+dx)*(d_jpool_e[n-2] - den);
        d_jpool_h[n-1] = dhm + (dt*LS-dx)/(dt*LS+dx)*(d_jpool_h[n-2] - dhn);

        //copybound
        d_jpvals[0] = d_jpool_e[0];
        d_jpvals[1] = d_jpool_e[1];
        d_jpvals[2] = d_jpool_h[0];
        d_jpvals[3] = d_jpool_h[1];
        d_jpvals[4] = d_jpool_e[n-1];
        d_jpvals[5] = d_jpool_e[n-2];
        d_jpvals[6] = d_jpool_h[n-1];
        d_jpvals[7] = d_jpool_h[n-2];
      
        //applysource
        d_jpool_e[0] = e;

        //ystep_h
        for (i=0; i<(n-1); i++)
           d_jpool_h[i] += 1.0/(d_jpool_mu[i])*dMulth*(d_jpool_e[i] - d_jpool_e[i+1]);  

}
*/

/*set of functions for */ 
/*
__global__  void
mbnxeKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
          int xres, int yres, int zres, int mb_bx0, int mb_bxn, 
          int mb_bx0pos, int mb_bxnpos, int mb_by0pos, int mb_bynpos, int mb_bz0pos, int mb_bznpos, int dir)
{
    int i, j, k, pos1, pos2;
    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    if ((mb_bx0 == 4 || mb_bxn == 4) && (i==mb_bx0pos && j>=mb_by0pos && j<mb_bynpos && k>=mb_bz0pos && k<mb_bznpos)) {
        pos1 = CAT(i, j, k);
        pos2 = CAT(mb_bxnpos, j, k);
        d_ey[pos2] = d_ey[pos1];
        d_ez[pos2] = d_ez[pos1];
     }
}
*/

/*
__global__  void
mbnyeKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
          int xres, int yres, int zres, int mb_by0, int mb_byn, 
          int mb_bx0pos, int mb_bxnpos, int mb_by0pos, int mb_bynpos, int mb_bz0pos, int mb_bznpos, int dir)
{
    int i, j, k, pos1, pos2;
    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    if ((mb_by0 == 4 || mb_byn == 4) && (j==mb_by0pos && i>=mb_bx0pos && i<mb_bxnpos && k>=mb_bz0pos && k<mb_bznpos)) {
        pos1 = CAT(i, j, k);
        pos2 = CAT(i, mb_bynpos, k);
        d_ex[pos2] = d_ex[pos1];
        d_ez[pos2] = d_ez[pos1];
    }
}
*/

/*
__global__  void
mbnxhKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
          int xres, int yres, int zres, int mb_bx0, int mb_bxn, 
          int mb_bx0pos, int mb_bxnpos, int mb_by0pos, int mb_bynpos, int mb_bz0pos, int mb_bznpos, int dir)
{
    int i, j, k, pos1, pos2;
    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    if ((mb_bx0 == 4 || mb_bxn == 4) && (i==mb_bx0pos && j>=mb_by0pos && j<mb_bynpos && k>=mb_bz0pos && k<mb_bznpos)) {
        pos1 = CAT(i-1, j, k);
        pos2 = CAT(mb_bxnpos-1, j, k);
        d_hy[pos1] = d_hy[pos2];
        d_hz[pos1] = d_hz[pos2];
     }
}
*/

/*
__global__  void
mbnyhKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
          int xres, int yres, int zres, int mb_by0, int mb_byn, 
          int mb_bx0pos, int mb_bxnpos, int mb_by0pos, int mb_bynpos, int mb_bz0pos, int mb_bznpos, int dir)
{
    int i, j, k, pos1, pos2;
    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    if ((mb_by0 == 4 || mb_byn == 4) && (j==mb_by0pos && i>=mb_bx0pos && i<mb_bxnpos && k>=mb_bz0pos && k<mb_bznpos)) {
        pos1 = CAT(i, j-1, k);
        pos2 = CAT(i, mb_bynpos-1, k);
        d_hx[pos1] = d_hx[pos2];
        d_hz[pos1] = d_hz[pos2];
     }

}
*/

/*

__device__ float
k_dcomp(int i, int j, int k, int xres, int yres, int zres, float theta, float phi,
      int i0, int i1, int j0, int j1, int k0, int k1)
{
        float rx, ry, rz;
        float ax, ay, az;

        ax = sin(theta)*cos(phi);
        ay = sin(theta)*sin(phi);
        az = cos(theta);
        if (theta >= 0 && theta <= (CU_PI/2.0)) {
            if (phi >= 0 && phi <= (CU_PI/2.0)) {
                rx = i - i0;
                ry = j - j0;
                rz = k - k0;
            }
            else if (phi > (CU_PI/2.0) && phi <= CU_PI) {
                rx = i - i1;
                ry = j - j0;
                rz = k - k0;
            }
            else if (phi > CU_PI && phi <= (3.0*CU_PI/2.0)) {
                rx = i - i1;
                ry = j - j1;
                rz = k - k0;
            }
            else {
                rx = i - i0;
                ry = j - j1;
                rz = k - k0;
            }
        }
        else if (theta < CU_PI && theta > (CU_PI/2.0)) 
        {
            if (phi >= 0 && phi <= (CU_PI/2.0)) {
                rx = i - i0;
                ry = j - j0;
                rz = k - k1;
            }
            else if (phi > (CU_PI/2.0) && phi <= CU_PI) {
                rx = i - i1;
                ry = j - j0;
                rz = k - k1;
            }
            else if (phi > CU_PI && phi <= (3.0*CU_PI/2.0)) {
                rx = i - i1;
                ry = j - j1;
                rz = k - k1;
            }
            else {
                rx = i - i0;
                ry = j - j1;
                rz = k - k1;
            }
         }
    return 10 + (ax*rx + ay*ry + az*rz);
}
*/

/*
__device__ float
k_rdcomp(int i, int j, int k, int xres, int yres, int zres, float theta, float phi,
      int i0, int i1, int j0, int j1, int k0, int k1)
{
        float rx, ry, rz;
        float ax, ay, az;

        ax = sin(theta)*cos(phi);
        ay = sin(theta)*sin(phi);
        az = cos(theta);

        rx = -i+xres/2;
        ry = -j+yres/2;
        rz = -k+zres/2;

        if ((ax*rx + ay*ry + az*rz)<250)
        return 250 - (ax*rx + ay*ry + az*rz);
        else return 0;
}
*/

/*
__device__ float k_gex(float field, float theta, float phi, float psi)
{
    return field*(cos(psi)*sin(phi) - sin(psi)*cos(theta)*cos(phi));
}
__device__ float k_gey(float field, float theta, float phi, float psi)
{
    return field*(-cos(psi)*cos(phi) - sin(psi)*cos(theta)*sin(phi));
}
__device__ float k_gez(float field, float theta, float phi, float psi)
{
    return field*(sin(psi)*sin(theta));
}
__device__ float k_ghx(float field, float theta, float phi, float psi)
{
    return field*(sin(psi)*sin(phi) + cos(psi)*cos(theta)*cos(phi));
}
__device__ float k_ghy(float field, float theta, float phi, float psi)
{
    return field*(-sin(psi)*cos(phi) + cos(psi)*cos(theta)*sin(phi));
}
__device__ float k_ghz(float field, float theta, float phi, float psi)
{
    return field*(-cos(psi)*sin(theta));
}
*/

/*
__device__ float
get_dval(float *line, int res, float x)
{
    float w1, w2, w3, w4;
    int l = (int)(x);
    float a = x-(float)l;

    if (x>=1 && x<(res-1))
    {
        w1=a+1; w2=a; w3=1-a; w4=2-a;
        w1=4-8*w1+5*w1*w1-w1*w1*w1;
        w2=1-2*w2*w2+w2*w2*w2;
        w3=1-2*w3*w3+w3*w3*w3;
        w4=4-8*w4+5*w4*w4-w4*w4*w4;
        return w1*line[l-1]+w2*line[l]+w3*line[l+1]+w4*line[l+2];
    } else if ((x<1 && x>=0) || x>=(res-1) && x<(res))
    {
        return (1-a)*line[l]+a*line[l+1];
    }
    else return 0;
}
*/
/*

__global__  void
tsf_estep_aKernel(float *d_ex, float *d_ey, float *d_ez,
          float *d_hx, float *d_hy, float *d_hz,
          float *d_epsilon, float *d_mu, int matmode,
          float *d_tsfset,
          float *d_jpool_e, float *d_jpool_h, int jpool_size,
          float dx, float dt,
          int xres, int yres, int zres, int dir)
{
    int i, j, k;
    long int pos;
    float d, vh;  
    int i0 = (int)d_tsfset[0];
    int j0 = (int)d_tsfset[1];
    int k0 = (int)d_tsfset[2];
    int i1 = (int)d_tsfset[3];
    int j1 = (int)d_tsfset[4];
    int k1 = (int)d_tsfset[5];
    float theta = d_tsfset[6];
    float phi = d_tsfset[7];
    float psi = d_tsfset[8];
    float corr = d_tsfset[9];
    float epsilon;
    float gcorr = 1;
    int skip_i0 = (int)d_tsfset[10];
    int skip_in = (int)d_tsfset[11];
    int gaussian = (int)d_tsfset[16];
    float gaussian_fxpos = d_tsfset[17];
    float gaussian_fypos = d_tsfset[18];
    float gaussian_rx = d_tsfset[19];
    float gaussian_ry = d_tsfset[20];

    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    pos = CAT(i, j, k);
    if (matmode == 0 || matmode == 1)
        epsilon = d_epsilon[pos];
    else epsilon = EPSILON_0;


    if ((!skip_i0) && i==(i0))
    {
        if (j>=j0 && j<=j1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i0-1, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i0-1, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             vh = gcorr*get_dval(d_jpool_h, jpool_size, d);

             if (j<j1) d_ey[pos] += dt/dx/epsilon*
                     k_ghz(vh, theta, phi, psi);                                         
             if (k<k1) d_ez[pos] -= dt/dx/epsilon*
                     k_ghy(vh, theta, phi, psi);                                         
        }
    }
    if ((!skip_in) && i==i1)
    {
        if (j>=j0 && j<=j1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i1, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             vh = gcorr*get_dval(d_jpool_h, jpool_size, d);

             if (j<j1) d_ey[pos] -= dt/dx/epsilon*
                     k_ghz(vh, theta, phi, psi);                                         
             if (k<k1) d_ez[pos] += dt/dx/epsilon*
                     k_ghy(vh, theta, phi, psi);                                         
        }
    }
}
*/

/*
__global__  void
tsf_estep_bKernel(float *d_ex, float *d_ey, float *d_ez,
          float *d_hx, float *d_hy, float *d_hz,
          float *d_epsilon, float *d_mu, int matmode,
          float *d_tsfset,
          float *d_jpool_e, float *d_jpool_h, int jpool_size,
          float dx, float dt,
          int xres, int yres, int zres, int dir)
{
    int i, j, k;
    long int pos;
    float d, vh, epsilon;    
    int i0 = (int)d_tsfset[0];
    int j0 = (int)d_tsfset[1];
    int k0 = (int)d_tsfset[2];
    int i1 = (int)d_tsfset[3];
    int j1 = (int)d_tsfset[4];
    int k1 = (int)d_tsfset[5];
    float theta = d_tsfset[6];
    float phi = d_tsfset[7];
    float psi = d_tsfset[8];
    float corr = d_tsfset[9];
    float gcorr = 1;
    int skip_j0 = (int)d_tsfset[12];
    int skip_jn = (int)d_tsfset[13];
    int gaussian = (int)d_tsfset[16];
    float gaussian_fxpos = d_tsfset[17];
    float gaussian_fypos = d_tsfset[18];
    float gaussian_rx = d_tsfset[19];
    float gaussian_ry = d_tsfset[20];

    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    pos = CAT(i, j, k);

    if (matmode == 0 || matmode == 1)
        epsilon = d_epsilon[pos];
    else epsilon = EPSILON_0;

    if ((!skip_j0) && j==(j0))
    {
        if (i>=i0 && i<=i1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j0-1, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j0-1, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             vh = gcorr*get_dval(d_jpool_h, jpool_size, d);

             if (i<i1) d_ex[pos] -= dt/dx/epsilon*
                     k_ghz(vh, theta, phi, psi);                                         
             if (k<k1) d_ez[pos] += dt/dx/epsilon*
                     k_ghx(vh, theta, phi, psi);                                         
        }
    }
    if ((!skip_jn) && j==j1)
    {
        if (i>=i0 && i<=i1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j1, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             vh = gcorr*get_dval(d_jpool_h, jpool_size, d);

             if (i<i1) d_ex[pos] += dt/dx/epsilon*
                     k_ghz(vh, theta, phi, psi);                                         
             if (k<k1) d_ez[pos] -= dt/dx/epsilon*
                     k_ghx(vh, theta, phi, psi);                                         
        }
    }

}
*/

/*
__global__  void
tsf_hstep_aKernel(float *d_ex, float *d_ey, float *d_ez,
          float *d_hx, float *d_hy, float *d_hz,
          float *d_epsilon, float *d_mu, int matmode,
          float *d_tsfset,
          float *d_jpool_e, float *d_jpool_h, int jpool_size,
          float dx, float dt,
          int xres, int yres, int zres, int dir)
{
    int i, j, k;
    long int pos;
    float d, ve, mu;  
    int i0 = (int)d_tsfset[0];
    int j0 = (int)d_tsfset[1];
    int k0 = (int)d_tsfset[2];
    int i1 = (int)d_tsfset[3];
    int j1 = (int)d_tsfset[4];
    int k1 = (int)d_tsfset[5];
    float theta = d_tsfset[6];
    float phi = d_tsfset[7];
    float psi = d_tsfset[8];
    float corr = d_tsfset[9];
    float gcorr = 1;
    int skip_i0 = (int)d_tsfset[10];
    int skip_in = (int)d_tsfset[11];
    int gaussian = (int)d_tsfset[16];
    float gaussian_fxpos = d_tsfset[17];
    float gaussian_fypos = d_tsfset[18];
    float gaussian_rx = d_tsfset[19];
    float gaussian_ry = d_tsfset[20];




    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    pos = CAT(i, j, k);
    if (matmode == 0 || matmode == 2)
        mu = d_mu[pos];
    else mu = MU_0;


    if ((!skip_i0) && i==i0)
    {
        if (j>=j0 && j<=j1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
             
             ve = gcorr*get_dval(d_jpool_e, jpool_size, d);

             if (k<k1) d_hy[CAT(i-1, j, k)] -= dt/dx/mu*
                     k_gez(ve, theta, phi, psi);                                         
             if (j<j1) d_hz[CAT(i-1, j, k)] += dt/dx/mu*
                     k_gey(ve, theta, phi, psi);            
        }
    }

    if ((!skip_in) && i==i1)
    {
        if (j>=j0 && j<=j1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             ve = gcorr*get_dval(d_jpool_e, jpool_size, d);

             if (k<k1) d_hy[pos] += dt/dx/mu*
                     k_gez(ve, theta, phi, psi);                                         
             if (j<j1) d_hz[pos] -= dt/dx/mu*
                     k_gey(ve, theta, phi, psi);            
        }
    }

}
*/
/*
__global__  void
tsf_hstep_bKernel(float *d_ex, float *d_ey, float *d_ez,
          float *d_hx, float *d_hy, float *d_hz, 
          float *d_epsilon, float *d_mu, int matmode,
          float *d_tsfset,
          float *d_jpool_e, float *d_jpool_h, int jpool_size,
          float dx, float dt,
          int xres, int yres, int zres, int dir)
{
    int i, j, k;
    long int pos;
    float d, ve, mu;    
    int i0 = (int)d_tsfset[0];
    int j0 = (int)d_tsfset[1];
    int k0 = (int)d_tsfset[2];
    int i1 = (int)d_tsfset[3];
    int j1 = (int)d_tsfset[4];
    int k1 = (int)d_tsfset[5];
    float theta = d_tsfset[6];
    float phi = d_tsfset[7];
    float psi = d_tsfset[8];
    float corr = d_tsfset[9];
    float gcorr = 1;
    int skip_j0 = (int)d_tsfset[12];
    int skip_jn = (int)d_tsfset[13];
    int gaussian = (int)d_tsfset[16];
    float gaussian_fxpos = d_tsfset[17];
    float gaussian_fypos = d_tsfset[18];
    float gaussian_rx = d_tsfset[19];
    float gaussian_ry = d_tsfset[20];



    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       k = threadIdx.x;  
    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

    pos = CAT(i, j, k);
    if (matmode == 0 || matmode == 2)
        mu = d_mu[pos];
    else mu = MU_0;

 

    if ((!skip_j0) && j==j0)
    {
        if (i>=i0 && i<=i1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             ve = gcorr*get_dval(d_jpool_e, jpool_size, d);

             if (k<k1) d_hx[CAT(i, j-1, k)] += dt/dx/mu*
                     k_gez(ve, theta, phi, psi);                                         
             if (i<i1) d_hz[CAT(i, j-1, k)] -= dt/dx/mu*
                     k_gex(ve, theta, phi, psi);            
        }
    }
    if ((!skip_jn) && j==j1)
    {
        if (i>=i0 && i<=i1 && k>=k0 && k<=k1)
        {
             if (gaussian) gcorr*=gaussmult(i, j, k, gaussian_fxpos, gaussian_fypos, gaussian_rx, gaussian_ry);
             d = corr*k_dcomp(i, j, k,
                          xres, yres, zres,
                          theta, phi,
                          0, xres, 0, yres, 0, zres);
              
             ve = gcorr*get_dval(d_jpool_e, jpool_size, d);

             if (k<k1) d_hx[pos] -= dt/dx/mu*
                     k_gez(ve, theta, phi, psi);                                         
             if (i<i1) d_hz[pos] += dt/dx/mu*
                     k_gex(ve, theta, phi, psi);            
        }
    }

}
*/


/*
__device__ void 
get_matprops(int i, int j, int k, float *epsilon, float *mu,
             int *d_mat, int *d_mattype, float *d_mattab, int matmode, int nmat,
             float *d_epsilon, float *d_mu, int xres, int yres, int zres)
{
	long int pos = CAT(i, j, k);
	int mattype = d_mattype[d_mat[pos]];  //0..normal, >1..tabulated

	if (matmode==0) {
		*epsilon = EPSILON_0;
		*mu = MU_0;
	} 
	else if (matmode == 1 || matmode==2 || matmode==3) { //= mattype 0 = linear material given pixel by pixel
		if (nmat==0 || d_mat[pos] == 0) {
			if (matmode == 2) *epsilon = EPSILON_0;
			else *epsilon = d_epsilon[pos];
			if (matmode == 1) *mu = MU_0;
			*mu = d_mu[pos];
		} else { //some tabulated material inside linear material
			*epsilon = d_mattab[GETEPS(d_mat[pos])]; 
			*mu = d_mattab[GETMU(d_mat[pos])];
		}
	} else if (nmat>0 && mattype==1) { //tabulated linear material
		*epsilon = d_mattab[GETEPS(d_mat[pos])]; 
		*mu = d_mattab[GETMU(d_mat[pos])];
	} else {
		*epsilon = EPSILON_0;
		*mu = MU_0;
	}

}
*/

/*
__global__  void
fastsumKernel_a(float *d_ex, float *d_ey, float *d_ez, int xres, int yres, int zres, 
          float *d_epsilon, float *d_mu, float *d_sum_epsilon, float *d_sum_sigma, int *d_sum_mode, 
          int *i0, int *i1, int *j0, int *j1, int *k0, int *k1,
          int nsums, float *d_acc, int dir,
          int *d_mat, int *d_mattype, float *d_mattab, int matmode, int nmat)
{
 
    int m, i, j, k, pos;
    float val, epsilon, mu;

    if (dir==2) {
       i = blockIdx.x;
       j = blockIdx.y;
       m = threadIdx.x;  

       d_acc[m*xres*yres + i*xres + j] = 0;

       if (i<i0[m] || i>=i1[m] || j<j0[m] || j>=j1[m]) return;

       
       for (k=k0[m]; k<k1[m]; k++)
       {
           pos = CAT(i, j, k);

           get_matprops(i, j, k, &epsilon, &mu, 
				    d_mat, d_mattype, d_mattab, matmode, nmat, d_epsilon, d_mu, xres, yres, zres);

            if (fabs(epsilon-d_sum_epsilon[m])<(EPSILON_0/100.0))
           {
               val = 0;
               if (d_sum_mode[m] == 0 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                   val += d_ex[pos]*d_ex[pos];
               if (d_sum_mode[m] == 1 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                   val += d_ey[pos]*d_ey[pos];
               if (d_sum_mode[m] == 2 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                   val += d_ez[pos]*d_ez[pos];

               if (d_sum_mode[m] == 4) d_acc[m*xres*yres + i*xres + j] += d_sum_sigma[m]*val;
               else d_acc[m*xres*yres + i*xres + j] += val;
           }
       }

    } else if (dir==0) {
       j = blockIdx.x;
       k = blockIdx.y;
       i = threadIdx.x;  
    } else {
       i = blockIdx.x;
       k = blockIdx.y;
       j = threadIdx.x;  
    }

}
*/

/*
__global__  void
fastsumKernel_b(float *d_acc, float *d_sums, int nsums, int nsteps, int step, int xres, int yres, int zres, int dir)
{
    int i, j;

    int m = threadIdx.x;
    
    if (dir==2) {
       for (i=0; i<xres; i++)
       {
          for (j=0; j<yres; j++)
          {
              d_sums[m*nsteps + step] += d_acc[m*xres*yres + i*xres + j];
          }
       }
    }
}
*/
  
/* 
__global__  void
sumKernel(float *d_ex, float *d_ey, float *d_ez, int xres, int yres, int zres, 
          float *d_epsilon, float *d_mu, float *d_sums, float *d_sum_epsilon, float *d_sum_sigma, int *d_sum_mode, 
          int *i0, int *i1, int *j0, int *j1, int *k0, int *k1,
          int nsteps, int step,
          int *d_mat, int *d_mattype, float *d_mattab, int matmode, int nmat)
{
    int i, j, k, pos;
    int m = threadIdx.x;
    float val, epsilon, mu;


    for (i=i0[m]; i<i1[m]; i++)
    {
        for (j=j0[m]; j<j1[m]; j++)
        {
            for (k=k0[m]; k<k1[m]; k++)
            {
                pos = CAT(i, j, k);
                val = 0;
    
                get_matprops(i, j, k, &epsilon, &mu, 
				    d_mat, d_mattype, d_mattab, matmode, nmat, d_epsilon, d_mu, xres, yres, zres);

                if (fabs(epsilon-d_sum_epsilon[m])<(EPSILON_0/100.0))
                {
                    if (d_sum_mode[m] == 0 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                        val += d_ex[pos]*d_ex[pos];
                    else if (d_sum_mode[m] == 1 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                        val += d_ey[pos]*d_ey[pos];
                    else if (d_sum_mode[m] == 2 || d_sum_mode[m] == 3 || d_sum_mode[m] == 4)
                        val += d_ez[pos]*d_ez[pos];

                    if (d_sum_mode[m] == 4) d_sums[m*nsteps + step] += d_sum_sigma[m]*val;
                    else d_sums[m*nsteps + step] += val;
                }
            }
        }
    }
}
*/




__global__  void
outpointKernel(float *d_ex, float *d_ey, float *d_ez, float *d_hx, float *d_hy, float *d_hz,
                              float *d_outpointdata, int *d_outpoint_pos, int nsteps, int noutpoints,
                              int step, int xres, int yres)
{
    int i;
    for (i=0; i<noutpoints; i++) 
    {
        d_outpointdata[6*(i*nsteps + step)] = d_ex[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
        d_outpointdata[6*(i*nsteps + step) + 1] = d_ey[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
        d_outpointdata[6*(i*nsteps + step) + 2] = d_ez[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
        d_outpointdata[6*(i*nsteps + step) + 3] = d_hx[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
        d_outpointdata[6*(i*nsteps + step) + 4] = d_hy[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
        d_outpointdata[6*(i*nsteps + step) + 5] = d_hz[CAT(d_outpoint_pos[3*i], d_outpoint_pos[3*i+1])];
    }
}



/*All the kernel wrappers called from rest of the code are here below*/
 
hipError_t wrap_hKernel(SvGpuPlan *plan)
{
        hipError_t err;

        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        if (plan->matmode == 0 || plan->matmode == 1)  {
 	   hKernel_none<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                                              plan->xres, plan->yres, plan->tmmode,
                                              plan->dx, plan->dy, plan->dt, plan->dir);
         }
        else 
   	   hKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                                              plan->d_epsilon, plan->d_mu, plan->d_sigma, plan->d_sigast, 
                                              plan->xres, plan->yres, plan->tmmode,
                                              plan->dx, plan->dy, plan->dt, plan->dir);
        
        err = hipGetLastError();
        if (err) printf("H returned \"%s\"\nH calling sync\n", hipGetErrorString(err));

        err = hipDeviceSynchronize();
        if (err) printf("H returned \"%s\"\n", hipGetErrorString(err));
        return hipGetLastError();
}

hipError_t wrap_eKernel(SvGpuPlan *plan)
{
        hipError_t err;

        //hipEvent_t evt;
        //hipEventCreate(&evt);
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        /*if (plan->h_isplrc && plan->step==0) {
             printf("Initialising plrc fields");
             plrcnullKernel<<<1, 1>>>(plan->d_plrcx, plan->d_plrcy, plan->d_plrcz,
                                     plan->xres, plan->yres, plan->zres);
        } */
      
        //printf("E Launch %d %d %d %g %g %g %g\n", plan->xres, plan->yres, plan->zres, plan->dx, plan->dy, plan->dz, plan->dt);

	
	if (plan->nmat>0) {
		eKernel_tab<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz, 
                                plan->d_epsilon, plan->d_mu, plan->d_sigma, plan->d_sigast,
				plan->d_mat, plan->d_mattype, plan->d_mattab, plan->nmat, plan->matmode,
				plan->xres, plan->yres, plan->tmmode,
				plan->dx, plan->dy, plan->dt, plan->dir);
	}
	else if (plan->matmode == 0 || plan->matmode == 2)  {
		eKernel_none<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz, 
				plan->xres, plan->yres, plan->tmmode,
				plan->dx, plan->dy, plan->dt, plan->dir);
	}
	else {
		eKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
				plan->d_epsilon, plan->d_mu, plan->d_sigma, plan->d_sigast, 
				plan->xres, plan->yres, plan->tmmode,
				plan->dx, plan->dy, plan->dt, plan->dir);
	}
	err = hipGetLastError();
//	hipEventRecord(evt, NULL);
//	while (hipEventQuery(evt) == hipErrorNotReady)
//	{
//		usleep(100);
//	}
//	hipEventDestroy(evt);

        if (err) printf("E returned \"%s\"\nE calling sync\n", hipGetErrorString(err));
        err = hipDeviceSynchronize();
        if (err) printf("E returned \"%s\"\n", hipGetErrorString(err));
        return hipGetLastError();
}
hipError_t wrap_liao_cpybnd(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        
	liaocpyKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                                              plan->d_liao_x0, plan->d_liao_xn, plan->d_liao_y0, plan->d_liao_yn, 
                                              plan->xres, plan->yres, 
                                              plan->d_bnds,
                                              plan->dx, plan->dy, plan->dt, plan->dir);

        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_liao_applybnd(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);
        

	liaorunKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz, plan->d_epsilon,
                                              plan->d_liao_x0, plan->d_liao_xn, plan->d_liao_y0, plan->d_liao_yn, 
                                              plan->xres, plan->yres, 
                                              plan->d_bnds, plan->matmode,
                                              plan->dx, plan->dy, plan->dt, plan->dir);

        hipDeviceSynchronize();
        return hipGetLastError();
}


hipError_t wrap_mbndx_e_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
  /*      mbnxeKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->xres, plan->yres, plan->zres, plan->mb_bx0, plan->mb_bxn, 
                              plan->mb_bx0pos, plan->mb_bxnpos, 
                              plan->mb_by0pos, plan->mb_bynpos,
                              plan->mb_bz0pos, plan->mb_bznpos, plan->dir);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_mbndy_e_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
  /*      mbnyeKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->xres, plan->yres, plan->zres, plan->mb_by0, plan->mb_byn, 
                              plan->mb_bx0pos, plan->mb_bxnpos, 
                              plan->mb_by0pos, plan->mb_bynpos,
                              plan->mb_bz0pos, plan->mb_bznpos, plan->dir);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_mbndz_e_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

       printf("Error! Not implemented yet.\n");
  /*      mbnzeKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->xres, plan->yres, plan->zres, plan->mb_bz0, plan->mb_bzn, 
                              plan->mb_bx0pos, plan->mb_bxnpos, 
                              plan->mb_by0pos, plan->mb_bynpos,
                              plan->mb_bz0pos, plan->mb_bznpos, plan->dir);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_mbndx_h_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
  /*      mbnxhKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->xres, plan->yres, plan->zres, plan->mb_bx0, plan->mb_bxn, 
                              plan->mb_bx0pos, plan->mb_bxnpos, 
                              plan->mb_by0pos, plan->mb_bynpos,
                              plan->mb_bz0pos, plan->mb_bznpos, plan->dir);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_mbndy_h_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
  /*      mbnyhKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->xres, plan->yres, plan->zres, plan->mb_by0, plan->mb_byn, 
                              plan->mb_bx0pos, plan->mb_bxnpos, 
                              plan->mb_by0pos, plan->mb_bynpos,
                              plan->mb_bz0pos, plan->mb_bznpos, plan->dir);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_srceKernel(SvGpuPlan *plan, int i, int j, int k, float ex, float ey, float ez)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        srcepointKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->xres, plan->yres, i, j, ex, ey, ez, plan->dir);
        hipDeviceSynchronize();
         
        return hipGetLastError();
}


hipError_t wrap_fastsumKernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
  /*      fastsumKernel_a<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->xres, plan->yres, plan->zres,
                                             plan->d_epsilon, plan->d_mu, plan->d_sum_epsilon, plan->d_sum_sigma, plan->d_sum_mode,
                                             plan->d_sum_i0, plan->d_sum_i1, plan->d_sum_j0, plan->d_sum_j1, plan->d_sum_k0, plan->d_sum_k1,
                                             plan->nsums, plan->d_sum_accumulator, plan->dir,
                                             plan->d_mat, plan->d_mattype, plan->d_mattab, plan->matmode, plan->nmat);
*/
        hipDeviceSynchronize();

        dim3 dimBlockb(plan->nsums);
        //fastsumKernel_b<<<1, dimBlockb>>>(plan->d_sum_accumulator, plan->d_sums, plan->nsums, plan->nsteps, plan->step, plan->xres, plan->yres, plan->zres,
        //                          plan->dir);

        hipDeviceSynchronize();
        return hipGetLastError();

}



hipError_t wrap_sumKernel(SvGpuPlan *plan)
{
        dim3 dimBlock(plan->nsums);

        printf("Error! Not implemented yet.\n");
  /*      sumKernel<<<1, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->xres, plan->yres, plan->zres, 
                                   plan->d_epsilon, plan->d_mu, plan->d_sums, plan->d_sum_epsilon, plan->d_sum_sigma, plan->d_sum_mode, 
                                   plan->d_sum_i0, plan->d_sum_i1, plan->d_sum_j0, plan->d_sum_j1, plan->d_sum_k0, plan->d_sum_k1,
                                   plan->nsteps, plan->step,
                                   plan->d_mat, plan->d_mattype, plan->d_mattab, plan->matmode, plan->nmat);
*/
        hipDeviceSynchronize();
         
        return hipGetLastError();
}


hipError_t wrap_srchKernel(SvGpuPlan *plan, int i, int j, int k, float hx, float hy, float hz)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        srchpointKernel<<<dimGrid, dimBlock>>>(plan->d_hx, plan->d_hy, plan->d_hz, plan->xres, plan->yres, i, j, hx, hy, hz, plan->dir);
        hipDeviceSynchronize();
         
        return hipGetLastError();
}

hipError_t wrap_ffKernel(SvGpuPlan *plan)
{
        dim3 dimBlock(plan->maxthreads);
        dim3 dimGrid((int)ceil((float)plan->h_iset[NPNTS]/plan->maxthreads));
        printf("Error! Not implemented yet.\n");
  /*      ffKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez,
                                        plan->d_iset, 
                                        plan->d_ff_ex, plan->d_ff_ey, plan->d_ff_ez,
                                        plan->xres, plan->yres, plan->zres,
                                        plan->dx, plan->dy, plan->dz, plan->dt, 
                                        plan->step, plan->maxthreads);
        
*/
        hipDeviceSynchronize();        
        return hipGetLastError();
}


hipError_t wrap_ffKernel_hlps(SvGpuPlan *plan)
{
        dim3 dimBlock(plan->h_iset[NPNTS]*plan->nhlps); //this should be 512 or 1024
        printf("Error! Not implemented yet.\n");
  /*      ffKernel_a<<<1, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez,
                                        plan->d_iset, 
                                        plan->d_ff_hlp_ex, plan->d_ff_hlp_ey, plan->d_ff_hlp_ez,
                                        plan->xres, plan->yres, plan->zres,
                                        plan->dx, plan->dy, plan->dz, plan->dt, 
                                        plan->step, plan->nhlps);
*/
        hipDeviceSynchronize();    
        return hipGetLastError();
}

hipError_t wrap_ffKernel_gethlps(SvGpuPlan *plan)
{
        printf("Error! Not implemented yet.\n");
  /*      ffKernel_b<<<1, 1>>>(plan->d_ff_ex, plan->d_ff_ey, plan->d_ff_ez,
                             plan->d_ff_hlp_ex, plan->d_ff_hlp_ey, plan->d_ff_hlp_ez,
                             plan->d_iset, plan->nhlps);*/
        hipDeviceSynchronize();    
        return hipGetLastError();
}



hipError_t wrap_outpointKernel(SvGpuPlan *plan)
{
        outpointKernel<<<1, 1>>>(plan->d_ex, plan->d_ey, plan->d_ez, plan->d_hx, plan->d_hy, plan->d_hz,
                              plan->d_outpointdata, plan->d_outpoint_pos, plan->nsteps, plan->noutpoints,
                              plan->step, plan->xres, plan->yres);              

        hipDeviceSynchronize();
        return hipGetLastError();


}

hipError_t wrap_tsf_jstepKernel(SvGpuPlan *plan, float e)
{
        printf("Error! Not implemented yet.\n");
  /*      tsfjstepKernel<<<1, 1>>>(plan->d_tsf_jpool_e, plan->d_tsf_jpool_h, plan->d_tsf_jpvals, 
                              plan->d_tsf_jpool_epsilon, plan->d_tsf_jpool_mu,
                              plan->d_tsf_jpool_sigma, plan->d_tsf_jpool_sigast,
                              plan->dx/plan->h_tsfset[9], plan->dt, e, plan->tsf_jpool_size);
*/
        hipDeviceSynchronize();
        return hipGetLastError();
}


hipError_t wrap_tsf_e_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
/*
        tsf_estep_aKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, 
                                         plan->d_hx, plan->d_hy, plan->d_hz,
                                         plan->d_epsilon, plan->d_mu, plan->matmode,
                                         plan->d_tsfset,
                                         plan->d_tsf_jpool_e, plan->d_tsf_jpool_h, plan->tsf_jpool_size,
                                         plan->dx, plan->dt,
                                         plan->xres, plan->yres, plan->zres, plan->dir);
        tsf_estep_bKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, 
                                         plan->d_hx, plan->d_hy, plan->d_hz,
                                         plan->d_epsilon, plan->d_mu, plan->matmode,
                                         plan->d_tsfset,
                                         plan->d_tsf_jpool_e, plan->d_tsf_jpool_h, plan->tsf_jpool_size,
                                         plan->dx, plan->dt,
                                         plan->xres, plan->yres, plan->zres, plan->dir);
*/

        hipDeviceSynchronize();
        return hipGetLastError();
}

hipError_t wrap_tsf_h_Kernel(SvGpuPlan *plan)
{
        int ga, b;

	b = plan->xres; //FIXME: do this dependent on problem size and graphics card computation capability
	ga = plan->yres;

	dim3 dimBlock(b);
        dim3 dimGrid(ga);

        printf("Error! Not implemented yet.\n");
/*
        tsf_hstep_aKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, 
                                         plan->d_hx, plan->d_hy, plan->d_hz,
                                         plan->d_epsilon, plan->d_mu, plan->matmode,
                                         plan->d_tsfset,
                                         plan->d_tsf_jpool_e, plan->d_tsf_jpool_h, plan->tsf_jpool_size,
                                         plan->dx, plan->dt,
                                         plan->xres, plan->yres, plan->zres, plan->dir);
        tsf_hstep_bKernel<<<dimGrid, dimBlock>>>(plan->d_ex, plan->d_ey, plan->d_ez, 
                                         plan->d_hx, plan->d_hy, plan->d_hz,
                                         plan->d_epsilon, plan->d_mu, plan->matmode,
                                         plan->d_tsfset,
                                         plan->d_tsf_jpool_e, plan->d_tsf_jpool_h, plan->tsf_jpool_size,
                                         plan->dx, plan->dt,
                                         plan->xres, plan->yres, plan->zres, plan->dir);
*/

        hipDeviceSynchronize();
        return hipGetLastError();
}


/*
__global__ void kernel() { ... }
hipError_t kernel_driver()
        {
        kernel<<<blocks, threads>>>();
        #ifdef NDEBUG
        return hipSuccess;
        #else
        hipDeviceSynchronize();
        return hipGetLastError();
        #endif
        }
*/


#endif

/* vim: set cin et ts=4 sw=4 cino=>1s,e0,n0,f0,{0,}0,^0,\:1s,=0,g1s,h0,t0,+1s,c3,(0,u0 : */
